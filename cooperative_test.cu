#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
namespace cg = cooperative_groups;

__global__ void test0()
{
  cg::grid_group gg = cg::this_grid();
  for(int j=0; j<100000; j++)
  {
    for(int i=0; i<100000; i++)
      asm volatile("nanosleep.u32 1000;");
    printf("execute0\n");
    gg.sync();
  }
  printf("finish0\n");
}
__global__ void test1()
{
  for(int j=0; j<100000; j++)
  {
    for(int i=0; i<100000; i++)
      asm volatile("nanosleep.u32 1000;");
    printf("execute1\n");
  }
  printf("finish1\n");
}


int main(int argc, char const *argv[])
{
  /* code */
  hipStream_t stream0;
  hipStreamCreate( &stream0 ); 
  hipStream_t stream1;
  hipStreamCreate( &stream1 ); 
// 2 cooperative
#ifdef TYPE0
  hipLaunchCooperativeKernel((void*)test0, 1, 1, NULL, 0,stream0);//<-Persistent Kernel Relies on it
  hipLaunchCooperativeKernel((void*)test1, 1, 1, NULL, 0,stream1);//<-Persistent Kernel Relies on it
#endif
//cooperative & traditinal
  hipLaunchCooperativeKernel((void*)test0, 1, 1, NULL, 0,stream0);//<-Persistent Kernel Relies on it
  test1<<<1,1,0,stream1>>>();
  hipDeviceSynchronize();
  hipStreamDestroy(stream0);
  hipStreamDestroy(stream1);
  return 0;
}